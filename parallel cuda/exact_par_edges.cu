#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <algorithm>
#include <climits>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "utils.h"
#include <vector>
#include <string>
#include <fstream>
#include <nlohmann/json.hpp>

using json = nlohmann::json;

#define CUDA_CHECK(err)                                                    \
  do                                                                       \
  {                                                                        \
    hipError_t err_ = (err);                                              \
    if (err_ != hipSuccess)                                               \
    {                                                                      \
      std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ \
                << ": " << hipGetErrorString(err_) << std::endl;          \
      exit(EXIT_FAILURE);                                                  \
    }                                                                      \
  } while (0)

const int MAX_N_IN_KERNEL = 32;

struct DFS_Task
{
  int start_node;
  int second_node;
};

struct DFS_State
{
  int current_node_val;
  int path_pos;
  int cur_l;
  int max_l;
  int child_idx_to_try;
};

__global__ void solve_kernel_decoupled_work(
    const int *d_graph,
    const bool *d_stop_vertices_check,
    int n,
    const DFS_Task *d_tasks,
    int num_tasks,
    int *d_min_max_l_shared_for_pruning,
    int *d_all_tasks_min_max_l,
    int *d_all_tasks_paths)
{
  int task_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (task_idx >= num_tasks)
  {
    return;
  }

  DFS_Task my_task = d_tasks[task_idx];
  int start_node_idx = my_task.start_node;
  int second_node_val = my_task.second_node;

  int current_path[MAX_N_IN_KERNEL];
  bool visited[MAX_N_IN_KERNEL];
  DFS_State dfs_stack[MAX_N_IN_KERNEL];
  int stack_top = -1;

  int thread_local_min_max_l_val = INT_MAX;
  int thread_local_best_path_arr[MAX_N_IN_KERNEL];

  for (int i = 0; i < n; ++i)
  {
    visited[i] = false;
    thread_local_best_path_arr[i] = -1;
  }

  current_path[0] = start_node_idx;
  current_path[1] = second_node_val;
  visited[start_node_idx] = true;
  visited[second_node_val] = true;

  int initial_edge_weight = d_graph[start_node_idx * n + second_node_val];
  int initial_cur_l = initial_edge_weight;
  int initial_max_l = 0;

  if (d_stop_vertices_check[second_node_val])
  {
    initial_max_l = initial_cur_l;
    initial_cur_l = 0;
  }

  dfs_stack[++stack_top] = {second_node_val, 2, initial_cur_l, initial_max_l, 0};

  int global_best_for_pruning_snapshot;

  while (stack_top != -1)
  {
    DFS_State u_state = dfs_stack[stack_top];
    global_best_for_pruning_snapshot = *d_min_max_l_shared_for_pruning;

    int effective_max_l = max(u_state.max_l, u_state.cur_l);
    int current_pruning_threshold = min(thread_local_min_max_l_val, global_best_for_pruning_snapshot);

    if (effective_max_l >= current_pruning_threshold && u_state.path_pos < n)
    {
      stack_top--;
      visited[u_state.current_node_val] = false;
      continue;
    }

    if (u_state.path_pos == n)
    {
      if (u_state.max_l < thread_local_min_max_l_val)
      {
        thread_local_min_max_l_val = u_state.max_l;
        for (int k = 0; k < n; ++k)
        {
          thread_local_best_path_arr[k] = current_path[k];
        }
        atomicMin(d_min_max_l_shared_for_pruning, thread_local_min_max_l_val);
      }
      stack_top--;
      visited[u_state.current_node_val] = false;
      continue;
    }

    bool found_child_to_explore = false;
    for (int v_node_candidate_idx = u_state.child_idx_to_try; v_node_candidate_idx < n; ++v_node_candidate_idx)
    {
      if (d_graph[u_state.current_node_val * n + v_node_candidate_idx] > 0 && !visited[v_node_candidate_idx])
      {
        dfs_stack[stack_top].child_idx_to_try = v_node_candidate_idx + 1;

        current_path[u_state.path_pos] = v_node_candidate_idx;
        visited[v_node_candidate_idx] = true;

        DFS_State v_state;
        v_state.current_node_val = v_node_candidate_idx;
        v_state.path_pos = u_state.path_pos + 1;
        v_state.child_idx_to_try = 0;

        int edge_weight = d_graph[u_state.current_node_val * n + v_node_candidate_idx];
        int accumulated_segment_len_at_v = u_state.cur_l + edge_weight;
        v_state.max_l = u_state.max_l;

        bool v_is_designated_stop = d_stop_vertices_check[v_node_candidate_idx];
        bool v_completes_hamiltonian_path = (v_state.path_pos == n);

        if (v_is_designated_stop || v_completes_hamiltonian_path)
        {
          v_state.max_l = max(v_state.max_l, accumulated_segment_len_at_v);
          v_state.cur_l = 0;
        }
        else
        {
          v_state.cur_l = accumulated_segment_len_at_v;
        }

        current_pruning_threshold = min(thread_local_min_max_l_val, global_best_for_pruning_snapshot);
        int v_effective_max_l = max(v_state.max_l, v_state.cur_l);

        if (v_effective_max_l >= current_pruning_threshold && !v_completes_hamiltonian_path)
        {
          visited[v_node_candidate_idx] = false;
          continue;
        }

        dfs_stack[++stack_top] = v_state;
        found_child_to_explore = true;
        break;
      }
    }

    if (!found_child_to_explore)
    {
      stack_top--;
      visited[u_state.current_node_val] = false;
    }
  }

  d_all_tasks_min_max_l[task_idx] = thread_local_min_max_l_val;
  if (thread_local_min_max_l_val != INT_MAX)
  {
    for (int k = 0; k < n; ++k)
    {
      d_all_tasks_paths[task_idx * n + k] = thread_local_best_path_arr[k];
    }
  }
}

int main(int argc, char **argv)
{
  if (argc < 2)
  {
    std::cerr << "Usage: " << argv[0] << " <path_to_json_data_file> [num_cuda_threads_per_block (optional, default 1024)]" << std::endl;
    return 1;
  }
  std::string test_data_path = argv[1];
  int threads_per_block = 1024;
  if (argc > 2)
  {
    try
    {
      threads_per_block = std::stoi(argv[2]);
      if (threads_per_block <= 0 || threads_per_block > 1024 || (threads_per_block & (threads_per_block - 1)) != 0)
      {
        std::cerr << "Threads per block must be > 0, <= 1024, and a power of 2. Using default 1024." << std::endl;
        threads_per_block = 1024;
      }
    }
    catch (const std::exception &e)
    {
      std::cerr << "Invalid argument for threads per block. Using default 1024. Error: " << e.what() << std::endl;
      threads_per_block = 1024;
    }
  }

  Utils utils;
  int n, s_num_designated_stops_in_list;
  int **h_graph_2d = nullptr;
  int *h_designated_stop_vertices_indices = nullptr;

  utils.read_data_from_json_to_arrays(test_data_path, n, s_num_designated_stops_in_list,
                                      h_graph_2d, h_designated_stop_vertices_indices);

  if (n == 0 || h_graph_2d == nullptr)
  {
    std::cout << "-1 (empty or invalid graph)" << std::endl;
    if (h_graph_2d)
      utils.release_allocated_memory(n, h_graph_2d, h_designated_stop_vertices_indices);
    return 0;
  }
  if (n > MAX_N_IN_KERNEL)
  {
    std::cerr << "Error: Number of vertices n=" << n
              << " exceeds MAX_N_IN_KERNEL=" << MAX_N_IN_KERNEL
              << ". Please recompile with a larger MAX_N_IN_KERNEL." << std::endl;
    utils.release_allocated_memory(n, h_graph_2d, h_designated_stop_vertices_indices);
    return 1;
  }

  std::vector<DFS_Task> h_tasks;
  for (int i = 0; i < n; ++i)
  {
    for (int j = 0; j < n; ++j)
    {
      if (i != j && h_graph_2d[i][j] > 0)
      {
        h_tasks.push_back({i, j});
      }
    }
  }

  if (h_tasks.empty())
  {
      std::cout << "-1 (no edges in graph, cannot form a path)" << std::endl;
      utils.release_allocated_memory(n, h_graph_2d, h_designated_stop_vertices_indices);
      return 0;
  }


  int *h_graph_flat = new int[n * n];
  for (int i = 0; i < n; ++i)
  {
    for (int j = 0; j < n; ++j)
    {
      h_graph_flat[i * n + j] = h_graph_2d[i][j];
    }
  }

  bool *h_stop_vertices_check = new bool[n]();
  for (int i = 0; i < s_num_designated_stops_in_list; ++i)
  {
    if (h_designated_stop_vertices_indices[i] >= 0 && h_designated_stop_vertices_indices[i] < n)
    {
      h_stop_vertices_check[h_designated_stop_vertices_indices[i]] = true;
    }
  }

  if (n > 0 && !utils.is_connected_arrays(n, h_graph_2d))
  {
    std::cout << "-2 (graph not connected)" << std::endl;
    delete[] h_graph_flat;
    delete[] h_stop_vertices_check;
    utils.release_allocated_memory(n, h_graph_2d, h_designated_stop_vertices_indices);
    return 0;
  }

  auto chrono_solve_start = std::chrono::high_resolution_clock::now();

  int *d_graph_flat;
  bool *d_stop_vertices_check_gpu;
  int *d_min_max_l_shared_for_pruning;
  DFS_Task *d_tasks;
  int *d_all_tasks_min_max_l;
  int *d_all_tasks_paths;

  int num_tasks = h_tasks.size();

  CUDA_CHECK(hipMalloc((void **)&d_graph_flat, n * n * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&d_stop_vertices_check_gpu, n * sizeof(bool)));
  CUDA_CHECK(hipMalloc((void **)&d_min_max_l_shared_for_pruning, sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&d_tasks, num_tasks * sizeof(DFS_Task)));
  CUDA_CHECK(hipMalloc((void **)&d_all_tasks_min_max_l, num_tasks * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&d_all_tasks_paths, num_tasks * n * sizeof(int)));

  CUDA_CHECK(hipMemcpy(d_graph_flat, h_graph_flat, n * n * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_stop_vertices_check_gpu, h_stop_vertices_check, n * sizeof(bool), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_tasks, h_tasks.data(), num_tasks * sizeof(DFS_Task), hipMemcpyHostToDevice));

  int h_initial_global_pruning_val = INT_MAX;
  CUDA_CHECK(hipMemcpy(d_min_max_l_shared_for_pruning, &h_initial_global_pruning_val, sizeof(int), hipMemcpyHostToDevice));

  std::vector<int> h_initial_min_max_l_for_tasks(num_tasks, INT_MAX);
  CUDA_CHECK(hipMemcpy(d_all_tasks_min_max_l, h_initial_min_max_l_for_tasks.data(), num_tasks * sizeof(int), hipMemcpyHostToDevice));

  int num_blocks = (num_tasks + threads_per_block - 1) / threads_per_block;

  std::cout << n << ' ';

  hipEvent_t kernel_start_event, kernel_stop_event;
  CUDA_CHECK(hipEventCreate(&kernel_start_event));
  CUDA_CHECK(hipEventCreate(&kernel_stop_event));
  CUDA_CHECK(hipEventRecord(kernel_start_event));

  solve_kernel_decoupled_work<<<num_blocks, threads_per_block>>>(
      d_graph_flat, d_stop_vertices_check_gpu, n,
      d_tasks, num_tasks,
      d_min_max_l_shared_for_pruning,
      d_all_tasks_min_max_l, d_all_tasks_paths);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipEventRecord(kernel_stop_event));
  CUDA_CHECK(hipEventSynchronize(kernel_stop_event));
  float kernel_milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&kernel_milliseconds, kernel_start_event, kernel_stop_event));

  std::vector<int> h_all_tasks_min_max_l(num_tasks);
  std::vector<int> h_all_tasks_paths(num_tasks * n);
  CUDA_CHECK(hipMemcpy(h_all_tasks_min_max_l.data(), d_all_tasks_min_max_l, num_tasks * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_all_tasks_paths.data(), d_all_tasks_paths, num_tasks * n * sizeof(int), hipMemcpyDeviceToHost));

  int global_best_min_max_l = INT_MAX;
  int best_task_idx = -1;

  for (int i = 0; i < num_tasks; ++i)
  {
    if (h_all_tasks_min_max_l[i] < global_best_min_max_l)
    {
      global_best_min_max_l = h_all_tasks_min_max_l[i];
      best_task_idx = i;
    }
  }
  
  auto chrono_solve_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> host_solve_duration_ms = chrono_solve_end - chrono_solve_start;

  if (best_task_idx == -1)
  {
    std::cout << "-1" << std::endl;
  }
  else
  {
    for (int i = 0; i < n; ++i)
    {
      std::cout << h_all_tasks_paths[best_task_idx * n + i] << (i == n - 1 ? "" : " ");
    }
    std::cout << " " << global_best_min_max_l;
    std::cout << " " << std::fixed << std::setprecision(3) << kernel_milliseconds;
    std::cout << " " << std::fixed << std::setprecision(3) << host_solve_duration_ms.count() << std::endl;
  }

  delete[] h_graph_flat;
  delete[] h_stop_vertices_check;
  utils.release_allocated_memory(n, h_graph_2d, h_designated_stop_vertices_indices);

  CUDA_CHECK(hipFree(d_graph_flat));
  CUDA_CHECK(hipFree(d_stop_vertices_check_gpu));
  CUDA_CHECK(hipFree(d_min_max_l_shared_for_pruning));
  CUDA_CHECK(hipFree(d_tasks));
  CUDA_CHECK(hipFree(d_all_tasks_min_max_l));
  CUDA_CHECK(hipFree(d_all_tasks_paths));
  CUDA_CHECK(hipEventDestroy(kernel_start_event));
  CUDA_CHECK(hipEventDestroy(kernel_stop_event));

  return 0;
}